#include "hip/hip_runtime.h"

#include <stdio.h>
#include <include/labwork.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>

#define ACTIVE_THREADS 4

int main(int argc, char **argv) {
    printf("USTHa aaaa ICT Master 2018, Advanced Programming for HPC.\n");
    if (argc < 2) {
        printf("Usage: labwork <lwNum> <inputImage>\n");
        printf("   lwNum        labwork number\n");
        printf("   inputImage   the input file name, in JPEG format\n");
        return 0;
    }

    int lwNum = atoi(argv[1]);
    std::string inputFilename;

    // pre-initialize CUDA to avoid incorrect profiling
    printf("Warming up...\n");
    char *temp;
    hipMalloc(&temp, 1024);

    Labwork labwork;
    if (lwNum != 2 ) {
        inputFilename = std::string(argv[2]);
        labwork.loadInputImage(inputFilename);
    }

    printf("Starting labwork %d\n", lwNum);
    Timer timer;
    timer.start();
    switch (lwNum) {
        case 1:
            labwork.labwork1_CPU();
            labwork.saveOutputImage("labwork2-cpu-out.jpg");
            printf("labwork 1 CPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            timer.start();
            labwork.labwork1_OpenMP();
            labwork.saveOutputImage("labwork2-openmp-out.jpg");
 	printf("labwork 1 OpenMP ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());

            break;
        case 2:
            labwork.labwork2_GPU();
            break;
        case 3:
            labwork.labwork3_GPU();
            labwork.saveOutputImage("labwork3-gpu-out.jpg");
   	    printf("labwork 3 GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            break;
        case 4:
            labwork.labwork4_GPU();
            labwork.saveOutputImage("labwork4-gpu-out.jpg");
 	printf("labwork 4 GPU ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            break;
        case 5:
	//    labwork.labwork5_CPU();
        //    labwork.saveOutputImage("labwork5-cpu-out.jpg");
            labwork.labwork5_GPU();
            labwork.saveOutputImage("labwork5-gpu-out.jpg");
            break;
        case 6:
            labwork.labwork6_GPU();
            labwork.saveOutputImage("labwork6-gpu-out.jpg");
            break;
        case 7:
            labwork.labwork7_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork7-gpu-out.jpg");
            break;
        case 8:
            labwork.labwork8_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork8-gpu-out.jpg");
            break;
        case 9:
            labwork.labwork9_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork9-gpu-out.jpg");
            break;
        case 10:
            labwork.labwork10_GPU();
            printf("[ALGO ONLY] labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
            labwork.saveOutputImage("labwork10-gpu-out.jpg");
            break;
    }
    printf("labwork %d ellapsed %.1fms\n", lwNum, timer.getElapsedTimeInMilliSec());
}

void Labwork::loadInputImage(std::string inputFileName) {
    inputImage = jpegLoader.load(inputFileName);
}

void Labwork::saveOutputImage(std::string outputFileName) {
    jpegLoader.save(outputFileName, outputImage, inputImage->width, inputImage->height, 90);
}

void Labwork::labwork1_CPU() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));
    for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }
    }
}

void Labwork::labwork1_OpenMP() {
    int pixelCount = inputImage->width * inputImage->height;
    outputImage = static_cast<char *>(malloc(pixelCount * 3));


    // do something here
#pragma omp parralel for
 for (int j = 0; j < 100; j++) {     // let's do it 100 times, otherwise it's too fast!
        for (int i = 0; i < pixelCount; i++) {
            outputImage[i * 3] = (char) (((int) inputImage->buffer[i * 3] + (int) inputImage->buffer[i * 3 + 1] +
                                          (int) inputImage->buffer[i * 3 + 2]) / 3);
            outputImage[i * 3 + 1] = outputImage[i * 3];
            outputImage[i * 3 + 2] = outputImage[i * 3];
        }


    }

}


int getSPcores(hipDeviceProp_t devProp) {
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major) {
        case 2: // Fermi
            if (devProp.minor == 1) cores = mp * 48;
            else cores = mp * 32;
            break;
        case 3: // Kepler
            cores = mp * 192;
            break;
        case 5: // Maxwell
            cores = mp * 128;
            break;
        case 6: // Pascal
            if (devProp.minor == 1) cores = mp * 128;
            else if (devProp.minor == 0) cores = mp * 64;
            else printf("Unknown device type\n");
            break;
        default:
            printf("Unknown device type\n");
            break;
    }
    return cores;
}

void Labwork::labwork2_GPU() {
    int nDevices = 0;
    // get all devices
    hipGetDeviceCount(&nDevices);
    printf("Number total of GPU : %d\n\n", nDevices);
    for (int i = 0; i < nDevices; i++){
        // get informations from individual device
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
	printf("%s\n", prop.name);
	printf("Clock Rate: %d\n", prop.clockRate);
	printf("Total global memory: %u", prop.totalGlobalMem);
	printf("Number of multiprocessors: %d\n", prop.multiProcessorCount);
	printf("Number of cores:     %d\n",getSPcores(prop) );
	printf("Warp size: %d\n", prop.warpSize);
	// something more here
    }

}

__global__ void grayscale2(uchar3 *input, uchar3 *output, int width, int height)
{
	int c = blockIdx.x*blockDim.x + threadIdx.x;
	int r = blockIdx.y*blockDim.y + threadIdx.y;
	int w=  width ;
	int tid = r*w + c;
	if (c<width)
	{
		if(r < height)
		{
			output[tid].x = (input[tid].x + input[tid].y + input[tid].z) / 3;
			output[tid].z = output[tid].y = output[tid].x;
		}
	}
}

__global__ void grayscale(uchar3 *input, uchar3 *output) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	output[tid].x = (input[tid].x + input[tid].y + input[tid].z) / 3;
	output[tid].z = output[tid].y = output[tid].x;
}
void Labwork::labwork3_GPU() {


    // Calculate number of pixels
        int pixelCount = inputImage->width * inputImage->height;
        outputImage = static_cast<char *>(malloc(pixelCount * 3));

    // Allocate CUDA memory
	uchar3 *devInput;
        uchar3 *devOutput;
        hipMalloc(&devInput, pixelCount * sizeof(uchar3));
        hipMalloc(&devOutput,pixelCount * sizeof(uchar3));

    // Copy CUDA Memory from CPU to GPU
	hipMemcpy(devInput, inputImage->buffer,pixelCount * sizeof(uchar3),hipMemcpyHostToDevice);

    // Processing
	int blockSize = 64;
	int numBlock = pixelCount / blockSize;
        printf("numblock %d\n", numBlock);
	grayscale<<<numBlock, blockSize>>>(devInput, devOutput);

    // Copy CUDA Memory from GPU to CPU
	hipMemcpy(outputImage, devOutput,pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);

    // Cleaning
	hipFree(devInput);
	hipFree(devOutput);
}
__global__ void blur(uchar3 *input, uchar3 *output, int width, int height)
{
	int convolution [7][7]={{0,0,1,2,1,0,0},{0,3,13,22,13,3,0},{1,13,59,97,59,13,1},{2,22,97,159,97,22,2},{1,13,59,97,59,13,1},{0,3,13,22,13,3,0},{0,0,1,2,1,0,0}};
        int tidx = blockIdx.x*blockDim.x + threadIdx.x;
        int tidy = blockIdx.y*blockDim.y + threadIdx.y;
	int tid = tidy*width+tidx;
	if (tidx<width)
        {
                if(tidy < height)
                {
			int left=0;
			int right=7;
			int up=0;
			int bottom=7;
			int somme=0;
			if (tidx-3<0)
			{
				left=3-tidx;
			}
			 if (tidy-3<0)
                        {
                                up=3-tidy;
                        }
			  if (width-tidx<3)
                        {
                                right=right-width+tidx;
                        }
			  if (height-tidy<3)
                        {
                                bottom=bottom-height+tidy;
                        }

			for (up= up ; up < bottom ; up++)
			{
				 for (left=left ; left < right ; left++)
	                        {
					somme=somme+(output[(tidy+up-3)*width + (tidx-3+left)].x)*convolution[up][left];
                	        }

			}
			int coeff=0;
			 for (int j =0; j < 7 ; j++)
                        {
                                 for (int i=0 ; i < 7 ; i++)
                                {
                                        coeff=coeff+convolution[j][i];
                                }

                        }
			output[tid].x= somme/coeff;
			output[tid].z = output[tid].y = output[tid].x;


                }
        }
}

void Labwork::labwork4_GPU() {
    // Calculate number of pixels
        int pixelCount = inputImage->width * inputImage->height;
        outputImage = static_cast<char *>(malloc(pixelCount * 3));

    // Allocate CUDA memory
        uchar3 *devInput;
        uchar3 *devOutput;
        hipMalloc(&devInput, pixelCount * sizeof(uchar3));
        hipMalloc(&devOutput,pixelCount * sizeof(uchar3));

    // Copy CUDA Memory from CPU to GPU
        hipMemcpy(devInput, inputImage->buffer,pixelCount * sizeof(uchar3),hipMemcpyHostToDevice);

    // Processing
	dim3 blockSize = dim3(32, 32);
	int width =inputImage->width / blockSize.x;
	int height=inputImage->height / blockSize.y;

	if ((inputImage->width % blockSize.x)>0)
	{
		width++;
	}
	if ((inputImage->height % blockSize.y)>0)
        {
		height++;
	}
	dim3 gridSize = dim3(width, height);
	grayscale2<<<gridSize, blockSize>>>(devInput, devOutput , inputImage->width, inputImage->height);

    // Copy CUDA Memory from GPU to CPU

        hipMemcpy(outputImage, devOutput,pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);

    // Cleaning
        hipFree(devInput);
        hipFree(devOutput);
}

void Labwork::labwork5_CPU() {
}

void Labwork::labwork5_GPU() {
  // Calculate number of pixels
        int pixelCount = inputImage->width * inputImage->height;
        outputImage = static_cast<char *>(malloc(pixelCount * 3));

    // Allocate CUDA memory
        uchar3 *devInput;
        uchar3 *devOutput;
        hipMalloc(&devInput, pixelCount * sizeof(uchar3));
        hipMalloc(&devOutput,pixelCount * sizeof(uchar3));
	uchar3 *devGray;
        hipMalloc(&devGray, pixelCount * sizeof(uchar3));

    // Copy CUDA Memory from CPU to GPU
        hipMemcpy(devInput, inputImage->buffer,pixelCount * sizeof(uchar3),hipMemcpyHostToDevice);

    // Processing
        dim3 blockSize = dim3(32, 32);
        int width =inputImage->width / blockSize.x;
        int height=inputImage->height / blockSize.y;

        if ((inputImage->width % blockSize.x)>0)
        {
                width++;
        }
        if ((inputImage->height % blockSize.y)>0)
        {
                height++;
        }
        dim3 gridSize = dim3(width, height);
        grayscale2<<<gridSize, blockSize>>>(devInput, devGray , inputImage->width, inputImage->height);

        blur<<<gridSize, blockSize>>>(devGray, devOutput , inputImage->width, inputImage->height);
    // Copy CUDA Memory from GPU to CPU


        hipMemcpy(outputImage, devOutput,pixelCount * sizeof(uchar3),hipMemcpyDeviceToHost);

    // Cleaning
        hipFree(devInput);
        hipFree(devOutput);

}

void Labwork::labwork6_GPU() {
}

void Labwork::labwork7_GPU() {
}

void Labwork::labwork8_GPU() {
}

void Labwork::labwork9_GPU() {

}

void Labwork::labwork10_GPU(){
}


























